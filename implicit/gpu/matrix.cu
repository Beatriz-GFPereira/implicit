
#include <hip/hip_runtime.h>
#include "hipblas.h"

#include "implicit/gpu/matrix.h"
#include "implicit/gpu/utils.cuh"

namespace implicit { namespace gpu {
template <typename T>
Vector<T>::Vector(int size, const T * host_data)
    : size(size) {
    CHECK_CUDA(hipMalloc(&data, size * sizeof(T)));
    if (host_data) {
        CHECK_CUDA(hipMemcpy(data, host_data, size * sizeof(T), hipMemcpyHostToDevice));
    }
}


template <typename T>
Vector<T>::~Vector() {
    CHECK_CUDA(hipFree(data));
}

template struct Vector<int>;
template struct Vector<float>;

Matrix::Matrix(int rows, int cols, float * host_data, bool cpu)
    : rows(rows), cols(cols) {
    if (cpu) {
        CHECK_CUDA(hipMalloc(&data, rows * cols * sizeof(float)));
        if (host_data) {
            CHECK_CUDA(hipMemcpy(data, host_data, rows * cols * sizeof(float), hipMemcpyHostToDevice));
        }
        owns_data = true;
    } else {
        data = host_data;
        owns_data = false;
    }
}

void Matrix::to_host(float * out) const {
    CHECK_CUDA(hipMemcpy(out, data, rows * cols * sizeof(float), hipMemcpyDeviceToHost));
}

Matrix::~Matrix() {
    if (owns_data) {
        CHECK_CUDA(hipFree(data));
    }
}

CSRMatrix::CSRMatrix(int rows, int cols, int nonzeros,
                             const int * indptr_, const int * indices_, const float * data_)
    : rows(rows), cols(cols), nonzeros(nonzeros) {

    CHECK_CUDA(hipMalloc(&indptr, (rows + 1) * sizeof(int)));
    CHECK_CUDA(hipMemcpy(indptr, indptr_, (rows + 1)*sizeof(int), hipMemcpyHostToDevice));

    CHECK_CUDA(hipMalloc(&indices, nonzeros * sizeof(int)));
    CHECK_CUDA(hipMemcpy(indices, indices_, nonzeros * sizeof(int), hipMemcpyHostToDevice));

    CHECK_CUDA(hipMalloc(&data, nonzeros * sizeof(float)));
    CHECK_CUDA(hipMemcpy(data, data_, nonzeros * sizeof(int), hipMemcpyHostToDevice));
}

CSRMatrix::~CSRMatrix() {
    CHECK_CUDA(hipFree(indices));
    CHECK_CUDA(hipFree(indptr));
    CHECK_CUDA(hipFree(data));
}

COOMatrix::COOMatrix(int rows, int cols, int nonzeros,
                             const int * row_, const int * col_, const float * data_)
    : rows(rows), cols(cols), nonzeros(nonzeros) {

    CHECK_CUDA(hipMalloc(&row, nonzeros * sizeof(int)));
    CHECK_CUDA(hipMemcpy(row, row_, nonzeros * sizeof(int), hipMemcpyHostToDevice));

    CHECK_CUDA(hipMalloc(&col, nonzeros * sizeof(int)));
    CHECK_CUDA(hipMemcpy(col, col_, nonzeros * sizeof(int), hipMemcpyHostToDevice));

    CHECK_CUDA(hipMalloc(&data, nonzeros * sizeof(float)));
    CHECK_CUDA(hipMemcpy(data, data_, nonzeros * sizeof(int), hipMemcpyHostToDevice));
}

COOMatrix::~COOMatrix() {
    CHECK_CUDA(hipFree(row));
    CHECK_CUDA(hipFree(col));
    CHECK_CUDA(hipFree(data));
}
}}  // namespace implicit::gpu
